/*
 * A + B = C vector summation
 * */
#include <iostream>
#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__
void vecAddKernel(float* A_d, float* B_d, float* C_d, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n) C_d[i] = A_d[i] + B_d[i];
}

int main(int argc, char *argv[]) {
  int n = atoi(argv[1]);
  cout << n << endl;

  size_t size = n * sizeof(float);

  // host memory
  float *a = (float *)malloc(size);
  float *b = (float *)malloc(size);
  float *c = (float *)malloc(size);

  for (int i = 0; i < n; ++i) {
    float af = rand() / double(RAND_MAX);
    float bf = rand() / double(RAND_MAX);
    a[i] = af;
    b[i] = bf;
  }

  // cuda memory
  float *da = NULL;
  float *db = NULL;
  float *dc = NULL;

  // This is a typecast. It indicates that the type of the pointer being passed is a pointer to void,
  // which is a generic type in C/C++ that can be used to represent any type.
  // In CUDA, memory allocation functions, like cudaMalloc, expect a pointer to void as their argument.
  hipMalloc((void **)&da, size);
  hipMalloc((void **)&db, size);
  hipMalloc((void **)&dc, size);

  hipMemcpy(da, a, size, hipMemcpyHostToDevice);
  hipMemcpy(db, a, size, hipMemcpyHostToDevice);
  hipMemcpy(dc, a, size, hipMemcpyHostToDevice);

  // kernel operate
  struct timeval t1, t2;

  int threadPerBlock = 256;
  gettimeofday(&t1, NULL);
  vecAddKernel<<<ceil(n / threadPerBlock), threadPerBlock>>>(da, db, dc, n);
  gettimeofday(&t2, NULL);

  // copy results to host
  hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

  double timeuse = (t2.tv_sec - t1.tv_sec) + (double)(t2.tv_usec - t1.tv_usec)/1000000.0;
  cout << timeuse << endl;

  hipFree(da);
  hipFree(db);
  hipFree(dc);

  free(a);
  free(b);
  free(c);
  return 0;
}